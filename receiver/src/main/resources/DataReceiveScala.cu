
#include <hip/hip_runtime.h>
#include <assert.h>

#define __assert(condition) \
  if (!(condition)) { return; }


extern "C"
// test reduce kernel that sums elements
__global__ void long_map_errcount(int n, long* in, long* out){
  //const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  if(ix<n){
    if(in[ix] > 100000)
      out[ix] = 0;
    else out[ix] = 1;
  }
}


extern "C"
__global__ void int_map_errcount(int n, int* in, int* out){
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  if(ix<n){
    if(in[ix]>=50 && in[ix]<80)
      out[ix] = 0;
    else
      out[ix] = 1;
  }
}

extern "C"
// test reduce kernel that sums elements
__global__ void sum(int *size, int *input, int *output, int *stage, int *totalStages) {
  const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
  const int jump = 64 * 256;
  // if (ix == 0) printf("size: %d stage : %d totalStages : %d \n",*size, *stage, *totalStages);
  if (*stage == 0) {
    if (ix < *size) {
      assert(jump == blockDim.x * gridDim.x);
      int result = 0;
      for (long i = ix; i < *size; i += jump) {
        result += input[i];
      }
      input[ix] = result;
    }
  } else if (ix == 0) {
    const long count = (*size < (long)jump) ? *size : (long)jump;
    int result = 0;
    for (long i = 0; i < count; ++i) {
      result += input[i];
    }
    output[0] = result;
  }
}


extern "C"
// test reduce kernel that sums elements
__global__ void suml(int size, long *input, long *output, int stage, int totalStages) {
  const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
  const int jump = 64 * 256;
  if (stage == 0) {
    if (ix < size) {
      assert(jump == blockDim.x * gridDim.x);
      long result = 0;
      for (long i = ix; i < size; i += jump) {
        result += input[i];
      }
      input[ix] = result;
    }
  } else if (ix == 0) {
    const long count = (size < (long)jump) ? size : (long)jump;
    long result = 0;
    for (long i = 0; i < count; ++i) {
      result += input[i];
    }
    output[0] = result;
  }
}


